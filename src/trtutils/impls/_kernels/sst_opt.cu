
#include <hip/hip_runtime.h>
#define TILE_DIM 32

extern "C" __global__
void scaleSwapTranspose_opt(
    const unsigned char* __restrict__ inImg,
    float* __restrict__ outImg,
    const float scale,
    const float offset,
    const int shape
) {
    // Declare shared memory statically for each channel.
    __shared__ float tileR[TILE_DIM][TILE_DIM + 1];
    __shared__ float tileG[TILE_DIM][TILE_DIM + 1];
    __shared__ float tileB[TILE_DIM][TILE_DIM + 1];

    // Compute the input coordinates
    int in_col = blockIdx.x * TILE_DIM + threadIdx.x;
    int in_row = blockIdx.y * TILE_DIM + threadIdx.y;

    // Load pixel from global memory if it's within the image bounds.
    if (in_row < shape && in_col < shape) {
        // Calculate the index for this pixel for interleaved BGR data.
        const int inputBaseIdx = (in_row * shape + in_col) * 3;
        // Perform scaling and offset conversion.
        float b = static_cast<float>(inImg[inputBaseIdx + 0]) * scale + offset;
        float g = static_cast<float>(inImg[inputBaseIdx + 1]) * scale + offset;
        float r = static_cast<float>(inImg[inputBaseIdx + 2]) * scale + offset;

        // Write the channels into shared memory.
        // The extra column (+1) avoids bank conflicts.
        tileR[threadIdx.y][threadIdx.x] = r;
        tileG[threadIdx.y][threadIdx.x] = g;
        tileB[threadIdx.y][threadIdx.x] = b;
    }

    // Synchronize to ensure the tile is fully loaded.
    __syncthreads();

    // Compute output coordinates - no transpose needed, output in same order as input
    int out_row = blockIdx.y * TILE_DIM + threadIdx.y;
    int out_col = blockIdx.x * TILE_DIM + threadIdx.x;

    if (out_row < shape && out_col < shape) {
        // Calculate the output base index in the planar (NCHW) format.
        int outIdx = out_row * shape + out_col;
        const int shapeSq = shape * shape;
        // Read from shared memory and write to output in RGB order (swap BGR to RGB)
        outImg[outIdx + 0 * shapeSq] = tileR[threadIdx.y][threadIdx.x];
        outImg[outIdx + 1 * shapeSq] = tileG[threadIdx.y][threadIdx.x];
        outImg[outIdx + 2 * shapeSq] = tileB[threadIdx.y][threadIdx.x];
    }
}
